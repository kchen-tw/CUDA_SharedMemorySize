﻿#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int sharedMemorySize;
    hipDeviceGetAttribute(&sharedMemorySize, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

    std::cout << "Maximum shared memory size per block: " << sharedMemorySize << " bytes" << std::endl;

    return 0;
}