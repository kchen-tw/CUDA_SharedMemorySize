﻿#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void kernel() {
    // 這是一個簡單的 CUDA 核心，可以是你的實際計算
}

int main() {
    int sharedMemorySize;
    
    hipDeviceGetAttribute(&sharedMemorySize, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

    cout << "Maximum shared memory size per block: " << sharedMemorySize << " bytes" << endl;

    int device;
    hipGetDevice(&device);

    int blockSize, minGridSize, gridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel, 0, 0);

    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, device);

    // 現在，你可以使用 minGridSize 和 blockSize 來計算 gridSize
    gridSize = (1 + (minGridSize - 1) / blockSize);

    cout << "Device: " << device << endl;
    cout << "Max Threads Per Block :" << maxThreadsPerBlock << endl;
    cout << "Maximum potential block size: " << blockSize << endl;
    cout << "Minimum grid size: " << minGridSize << endl;
    cout << "Computed grid size: " << gridSize << endl;


    return 0;
}